#include "hip/hip_runtime.h"
#include "holder_kernels.h"
#include "kernel/compact_kernel.cu"

namespace MEM
{
namespace MISC
{
__global__ void stretch_kernel( unsigned *in_data, unsigned *out_data, unsigned in_data_size, unsigned factor )
{
	unsigned index = threadIdx.x + blockDim.x * blockIdx.x;
	if( index >= in_data_size )
	{
		return;
	}
	unsigned r_factor = factor; // potrzebne? czy argumenty kernela już są w rejestrach?
	for( unsigned i = 0; i < r_factor; ++i )
	{
		out_data[ r_factor * index + i ] = in_data[ index ];
	}
}

void stretch( MEM::MISC::BufferCu<unsigned> *in, MEM::MISC::BufferCu<unsigned> *out, unsigned factor )
{
	unsigned threads = in->getLen();
	out->resize( threads * factor );
	dim3 block( min( 512, threads ) );
	dim3 grid( 1 + ( threads - 1 ) / block.x );
	stretch_kernel<<<grid, block>>>(
		in->d_data(),
		out->d_data(),
		threads,
		factor );
	CUT_CHECK_ERROR("kernel launch");
}

size_t reassign( void* data, BufferCu<unsigned> *indices, BufferCu<unsigned> *mask )
{
	unsigned size = indices->getLen();
	BufferCu<size_t> count(1);
	BufferCu<unsigned> tmp( size );
	unsigned threads = 1 + size / 8;
	dim3 block( min( 512, threads ) );
	dim3 grid( 1 + ( threads - 1 ) / block.x );
	compactData<unsigned, false><<<grid, block>>>(
		tmp.d_data(),
		count.d_data(),
		indices->d_data(),
		mask->d_data(),
		(unsigned*)data,
		size );
	CUT_CHECK_ERROR( "kernel launch" );
	hipMemcpy( data, tmp.d_data(), size * sizeof(unsigned), hipMemcpyDeviceToDevice );
	return count.retrieve();
}

} // namespace MISC
} // namespace MEM
