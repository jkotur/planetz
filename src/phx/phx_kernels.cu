#include "hip/hip_runtime.h"
#include "phx_kernels.h"

__device__ const float dt = 1e-3f;

__device__ inline float3 operator-( const float3 &l , const float3 &r )
{
	return make_float3( l.x - r.x, l.y - r.y, l.z - r.z );
}

__device__ inline float3 operator/( const float3 &v , const float &f )
{
	return make_float3( v.x / f, v.y / f, v.z / f );
}

__device__ inline float3 operator*( const float3 &v , const float &f )
{
	return make_float3( v.x * f, v.y * f, v.z * f );
}

__device__ inline float3& operator+=( float3& l, const float3& r )
{
	l.x += r.x;
	l.y += r.y;
	l.z += r.z;
	return l;
}

__device__ float3 get_dV( float3 myPos, float3 theirPos, float theirMass )
{
	float3 dir = theirPos - myPos;
	float r2 = dir.x * dir.x + dir.y * dir.y + dir.z * dir.z;
	return dir * (dt / ( r2 * sqrtf( r2 ) + 1e-3f) );
}

__global__ void basic_interaction( float3 *positions, float *masses, float3 *velocities, unsigned *cnt )
{
	unsigned index = blockIdx.x * blockDim.x + threadIdx.x;

	unsigned count = *cnt;
	if( index >= count )
	{
		return;
	}
	float3 new_pos = positions[ index ];
	float3 new_vel = velocities[ index ];

	for( unsigned i = 0; i < count; ++i )
	{
		if( i != index )
		{
			new_vel += get_dV( positions[ index ], positions[ i ], masses[ i ] );
			new_pos += new_vel * dt;
		}
	}

	__syncthreads();

	positions[ index ] = new_pos;
	velocities[ index ] = new_vel;
}
