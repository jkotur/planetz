#include "hip/hip_runtime.h"
#include <limits>
#include "phx.h"
#include "phx_kernels.h"
#include "phx_templates.h"
#include "kmeans.h"
#include "cuda/math.h"

using namespace PHX;

ConstChecker<float3, MEM::MISC::BufferGl> pos_checker;
ConstChecker<float, MEM::MISC::BufferCu> mass_checker;
ConstChecker<float3, MEM::MISC::BufferCu> vel_checker;

class Phx::CImpl
{
	public:
		CImpl(MEM::MISC::PhxPlanetFactory *p);
		virtual ~CImpl();

		void compute(unsigned n);
		void enableClusters(bool orly);
		bool clustersEnabled() const;

	private:
		void map_buffers();
		void unmap_buffers();

		void run_nbodies( unsigned planet_count );
		void run_nbodies2();
		void run_clusters();

		MEM::MISC::PhxPlanetFactory *planets;
		Clusterer clusterer;

		MEM::MISC::BufferCu<float3> tmp_pos;
		MEM::MISC::BufferCu<float3> tmp_vel;
		MEM::MISC::BufferCu<float> tmp_mass;

		bool clusters_on;
};

Phx::CImpl::CImpl(MEM::MISC::PhxPlanetFactory *p)
	: planets(p)
	, clusterer( &p->getPositions(), &p->getMasses() )
	, clusters_on( true )
{
}

Phx::CImpl::~CImpl()
{
}

void Phx::CImpl::compute(unsigned n)
{
	unsigned planet_count;
	if( !(planet_count = planets->size()) )
		return;
	map_buffers();
	for(unsigned i = 0; i < n; ++i)
	{
		vel_checker.setBuf( &planets->getVelocities() );
		mass_checker.setBuf( &planets->getMasses() );
		pos_checker.setBuf( &planets->getPositions() );
		run_clusters();
		pos_checker.checkBuf();
		mass_checker.checkBuf();
		vel_checker.checkBuf();
		run_nbodies( planet_count );
	}
	unmap_buffers();
}

void Phx::CImpl::map_buffers()
{
	planets->getPositions().map( MEM::MISC::BUF_CU );
	planets->getRadiuses().map( MEM::MISC::BUF_CU );
	planets->getCount().map( MEM::MISC::BUF_CU );
	tmp_pos.resize( planets->size() );
	tmp_vel.resize( planets->size() );
	hipMemset( tmp_pos.d_data(), 0, planets->size() * sizeof(float3) );
	hipMemset( tmp_vel.d_data(), 0, planets->size() * sizeof(float3) );
}

void Phx::CImpl::unmap_buffers()
{
	planets->getPositions().unmap();
	planets->getRadiuses().unmap();
	planets->getCount().unmap();
}
void Phx::CImpl::run_nbodies2()
{
	unsigned clusters = clusterer.getCount();
	unsigned *h_counts = new unsigned[ clusters ];
	clusterer.getCounts()->bind();
	memcpy( h_counts, clusterer.getCounts()->h_data(), clusters * sizeof(unsigned) );
	clusterer.getCounts()->unbind();
#ifdef PHX_DEBUG
	MEM::MISC::BufferCu<unsigned> whois( planets->size() );
	hipMemset( whois.d_data(), 0, planets->size() * sizeof(unsigned) );
#endif

	for( unsigned c = 0, prev_count = 0; c < clusters; ++c ) // TODO: odpalić te kernele jednocześnie?
	{
#ifdef PHX_DEBUG
		log_printf( DBG, "counts[%u]: %u\n", c, h_counts[c] );
#endif
		unsigned threads = h_counts[c] - prev_count;
		if( threads == 0 )
			continue;
#ifdef PHX_DEBUG
	float3 *d_dvs;
	hipMalloc( &d_dvs, threads * sizeof(float3) );
#endif
		dim3 block( min( threads, 512 ) );
		dim3 grid( 1 + ( threads - 1 ) / block.x );
		inside_cluster_interaction<<<grid, block>>>(
			planets->getPositions().map(MEM::MISC::BUF_CU),
			planets->getMasses().d_data(),
			planets->getVelocities().d_data(),
			clusterer.getShuffle()->d_data(),
			clusterer.getCounts()->d_data(),
			c, // cluster id
			tmp_pos.d_data(),
			tmp_vel.d_data()
#ifdef PHX_DEBUG
			, d_dvs, 1,whois.d_data()
#endif
			);
		CUT_CHECK_ERROR("Kernel launch");
		prev_count = h_counts[c];
#ifdef PHX_DEBUG
	float3 *dvs = new float3[ threads ];
	hipMemcpy( dvs, d_dvs, threads * sizeof(float3), hipMemcpyDeviceToHost );
	float3 sum_dvs = make_float3(0,0,0);
	for( unsigned i = 0; i < threads; ++i ) sum_dvs += dvs[i];
	std::string err = getErr();
	if( !err.empty() )
	{
		log_printf( _ERROR, "CUDA assertion failed: '%s'\n", err.c_str() );
		NOENTRY();
	}
	delete[] dvs;
#endif
	}
	hipMemcpy(
		planets->getPositions().map(MEM::MISC::BUF_CU),
		tmp_pos.d_data(), planets->size() * sizeof(float3), hipMemcpyDeviceToDevice );
	hipMemcpy(
		planets->getVelocities().d_data(),
		tmp_vel.d_data(), planets->size() * sizeof(float3), hipMemcpyDeviceToDevice );
#ifdef PHX_DEBUG
//	PRINT_OUT_BUF( whois, "%u" );
#endif
	delete[] h_counts;
}

void Phx::CImpl::run_nbodies( unsigned threads )
{	
	ASSERT( threads );
	if( clusters_on )
	{
		run_nbodies2();
		return;
	}
	dim3 block( min( threads, 512 ) );
	dim3 grid( 1 + (threads - 1) / block.x );
	//unsigned mem = block.x * ( sizeof(float3) + sizeof(float) );

#ifdef PHX_DEBUG
	float3 *d_dvs;
	hipMalloc( &d_dvs, threads * sizeof(float3) );
#endif
	basic_interaction<<<grid, block>>>( 
		planets->getPositions().map(MEM::MISC::BUF_CU), 
		planets->getMasses().d_data(), 
		planets->getVelocities().d_data(),
		planets->getCount().map(MEM::MISC::BUF_CU),
		tmp_pos.d_data(),
		tmp_vel.d_data()
#ifdef PHX_DEBUG
		, d_dvs, 4210
#endif
		);
	
	CUT_CHECK_ERROR("Kernel launch");
	
	
	hipMemcpy( planets->getPositions().map(MEM::MISC::BUF_CU), tmp_pos.d_data(), threads * sizeof(float3), hipMemcpyDeviceToDevice );
	hipMemcpy( planets->getVelocities().d_data(), tmp_vel.d_data(), threads * sizeof(float3), hipMemcpyDeviceToDevice );
#ifdef PHX_DEBUG
	float3 *dvs = new float3[ threads ];
	hipMemcpy( dvs, d_dvs, threads * sizeof(float3), hipMemcpyDeviceToHost );
	std::string err = getErr();
	if( !err.empty() )
	{
		log_printf( _ERROR, "CUDA assertion failed: '%s'\n", err.c_str() );
		NOENTRY();
	}
	delete[] dvs;
#endif
}

void Phx::CImpl::run_clusters()
{
	if( clusters_on )
	{
		clusterer.kmeans();
	}
}

void Phx::CImpl::enableClusters(bool orly)
{
	clusters_on = orly;
}

bool Phx::CImpl::clustersEnabled() const
{
	return clusters_on;
}

Phx::Phx(MEM::MISC::PhxPlanetFactory *p)
	: impl( new CImpl(p) )
{
}

Phx::~Phx()
{
	delete impl;
}

void Phx::compute(unsigned n)
{
	impl->compute(n);
}

void Phx::enableClusters(bool orly)
{
	impl->enableClusters(orly);
}

bool Phx::clustersEnabled() const
{
	return impl->clustersEnabled();
}
