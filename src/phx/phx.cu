#include "hip/hip_runtime.h"
#include <limits>
#include "phx.h"
#include "phx_kernels.h"
#include "phx_templates.h"
#include "kmeans.h"
#include "cuda/math.h"

using namespace PHX;

ConstChecker<float3, MEM::MISC::BufferGl> pos_checker;
ConstChecker<float, MEM::MISC::BufferCu> mass_checker;
ConstChecker<float3, MEM::MISC::BufferCu> vel_checker;

class Phx::CImpl
{
	public:
		CImpl(MEM::MISC::PhxPlanetFactory *p);
		virtual ~CImpl();

		void compute(unsigned n);
		void enableClusters(bool orly);
		bool clustersEnabled() const;

	private:
		void map_buffers();
		void unmap_buffers();

		void run_nbodies( unsigned planet_count );
		void run_nbodies2();
		void run_nbodies_for_clusters();
		void run_clusters();

		void update_positions();
		void handle_collisions();

		MEM::MISC::PhxPlanetFactory *planets;
		Clusterer clusterer;

		MEM::MISC::BufferCu<unsigned> merges1;
		MEM::MISC::BufferCu<unsigned> merges2;
		MEM::MISC::BufferCu<float3> tmp_vel;
		MEM::MISC::BufferCu<unsigned> filter; // a może użyć jakiegoś merge'a? taniej dla ramu
		bool clusters_on;
};

Phx::CImpl::CImpl(MEM::MISC::PhxPlanetFactory *p)
	: planets( p )
	, clusterer( p )
	, clusters_on( true )
{
}

Phx::CImpl::~CImpl()
{
}

void Phx::CImpl::compute(unsigned n)
{
	unsigned planet_count;
	if( !(planet_count = planets->size()) )
		return;
	map_buffers();
	for(unsigned i = 0; i < n; ++i)
	{
		vel_checker.setBuf( &planets->getVelocities() );
		mass_checker.setBuf( &planets->getMasses() );
		pos_checker.setBuf( &planets->getPositions() );
		run_clusters();
		pos_checker.checkBuf();
		mass_checker.checkBuf();
		vel_checker.checkBuf();
		run_nbodies( planet_count );
		update_positions();
	}
	handle_collisions();
	unmap_buffers();
}

void Phx::CImpl::map_buffers()
{
	planets->getPositions().map( MEM::MISC::BUF_CU );
	planets->getRadiuses().map( MEM::MISC::BUF_CU );
	planets->getCount().map( MEM::MISC::BUF_CU );
	merges1.resize( planets->size() );
	merges2.resize( planets->size() );
	tmp_vel.resize( planets->size() ); // prawdopodobnie wystarczy mniej, bo teraz to już tylko dla klastrów jest
}

void Phx::CImpl::unmap_buffers()
{
	planets->getPositions().unmap();
	planets->getRadiuses().unmap();
	planets->getCount().unmap();
}
void Phx::CImpl::run_nbodies2()
{
	unsigned clusters = clusterer.getCount();
	unsigned *h_counts = new unsigned[ clusters ];
	clusterer.getCounts()->bind();
	memcpy( h_counts, clusterer.getCounts()->h_data(), clusters * sizeof(unsigned) );
	clusterer.getCounts()->unbind();
#ifdef PHX_DEBUG
	MEM::MISC::BufferCu<unsigned> whois( planets->size() );
	hipMemset( whois.d_data(), 0, planets->size() * sizeof(unsigned) );
#endif

	for( unsigned c = 0, prev_count = 0; c < clusters; ++c ) // TODO: odpalić te kernele jednocześnie?
	{
		unsigned threads = h_counts[c] - prev_count;
		if( threads == 0 )
			continue;
#ifdef PHX_DEBUG
	float3 *d_dvs;
	hipMalloc( &d_dvs, threads * sizeof(float3) );
#endif
		dim3 block( min( threads, 512 ) );
		dim3 grid( 1 + ( threads - 1 ) / block.x );
		inside_cluster_interaction<<<grid, block>>>(
			planets->getPositions().map(MEM::MISC::BUF_CU),
			planets->getMasses().d_data(),
			planets->getVelocities().d_data(),
			clusterer.getShuffle()->d_data(),
			clusterer.getCounts()->d_data(),
			c // cluster id
#ifdef PHX_DEBUG
			, d_dvs, 1,whois.d_data()
#endif
			);
		CUT_CHECK_ERROR("Kernel launch");
		prev_count = h_counts[c];
#ifdef PHX_DEBUG
	float3 *dvs = new float3[ threads ];
	hipMemcpy( dvs, d_dvs, threads * sizeof(float3), hipMemcpyDeviceToHost );
	float3 sum_dvs = make_float3(0,0,0);
	for( unsigned i = 0; i < threads; ++i ) sum_dvs += dvs[i];
	std::string err = getErr();
	if( !err.empty() )
	{
		log_printf( _ERROR, "CUDA assertion failed: '%s'\n", err.c_str() );
		NOENTRY();
	}
	delete[] dvs;
#endif
	}
	delete[] h_counts;
}

void Phx::CImpl::run_nbodies( unsigned threads )
{	
	ASSERT( threads );
	if( clusters_on )
	{
		run_nbodies2();
		run_nbodies_for_clusters();
		return; // taaa, brzydkie, kiedyś będzie ładniej
	}
	dim3 block( min( threads, 512 ) );
	dim3 grid( 1 + (threads - 1) / block.x );
	//unsigned mem = block.x * ( sizeof(float3) + sizeof(float) );

#ifdef PHX_DEBUG
	float3 *d_dvs;
	hipMalloc( &d_dvs, threads * sizeof(float3) );
#endif
	basic_interaction<<<grid, block>>>( 
		planets->getPositions().map(MEM::MISC::BUF_CU), 
		planets->getMasses().d_data(), 
		planets->getVelocities().d_data(),
		planets->getCount().map(MEM::MISC::BUF_CU)
#ifdef PHX_DEBUG
		, d_dvs, 4210
#endif
		);
	
	CUT_CHECK_ERROR("Kernel launch");
	
#ifdef PHX_DEBUG
	float3 *dvs = new float3[ threads ];
	hipMemcpy( dvs, d_dvs, threads * sizeof(float3), hipMemcpyDeviceToHost );
	std::string err = getErr();
	if( !err.empty() )
	{
		log_printf( _ERROR, "CUDA assertion failed: '%s'\n", err.c_str() );
		NOENTRY();
	}
	delete[] dvs;
#endif
}

void Phx::CImpl::run_nbodies_for_clusters()
{
	unsigned threads = clusterer.getCount();
	ASSERT( threads <= 512 );
	dim3 block( min( threads, 512 ) );
	dim3 grid( 1 );

	static unsigned print_modulo = 0;
	print_modulo = (print_modulo+1)%5000;
	MEM::MISC::BufferCu<float3> *centers = clusterer.getCenters();
	centers->bind();
	if( print_modulo == 0 )
	for( unsigned i = 0; i < centers->getLen(); ++i )
	{
		log_printf( DBG, "centers[%u] = (%f,%f,%f)\n", i, centers->h_data()[i].x, centers->h_data()[i].y, centers->h_data()[i].z );
	}
	centers->unbind();
	outside_cluster_interaction<<<grid, block>>>(
		clusterer.getCenters()->d_data(),
		clusterer.getMasses()->d_data(),
		threads,
		tmp_vel.d_data() );
	CUT_CHECK_ERROR( "kernel launch" );

	tmp_vel.bind();
	if( print_modulo == 0 )
	for( unsigned i = 0; i < centers->getLen(); ++i )
	{
		log_printf( DBG, "vel[%u] = (%f, %f, %f)\n", i, tmp_vel.h_data()[i].x, tmp_vel.h_data()[i].y, tmp_vel.h_data()[i].z );
	}
	tmp_vel.unbind();

	threads = planets->size();
	block = min( threads, 512 );
	grid = 1 + ( threads - 1 ) / block.x;

	propagate_velocities<<<grid, block>>>(
		tmp_vel.d_data(),
		planets->getPositions().map(MEM::MISC::BUF_CU),
		planets->getVelocities().d_data(),
		clusterer.getShuffle()->d_data(),
		clusterer.getCounts()->d_data(),
		clusterer.getCount() - 1
		);
	CUT_CHECK_ERROR( "kernel launch" );
}

void Phx::CImpl::run_clusters()
{
	if( clusters_on )
	{
		clusterer.kmeans();
	}
}

void Phx::CImpl::update_positions()
{
	unsigned threads = planets->size();
	dim3 block( min( 512, threads ) );
	dim3 grid( 1 + ( threads - 1 ) / block.x );

	update_positions_kernel<<<grid, block>>>(
		planets->getPositions().map(MEM::MISC::BUF_CU),
		planets->getVelocities().d_data(),
		planets->getCount().map(MEM::MISC::BUF_CU) );
	CUT_CHECK_ERROR( "kernel launch" );
}

void Phx::CImpl::handle_collisions()
{
	bool merge_was_needed = false;
	MEM::MISC::BufferCu<unsigned> merge_needed(1);
	unsigned *in_merges = merges1.d_data();
	unsigned *out_merges = merges2.d_data();

	do
	{
		merge_needed.assign(0);
		unsigned threads = planets->size();
		dim3 block( min( 512, threads ) );
		dim3 grid( 1 + ( threads - 1 ) / block.x );
		
		detect_collisions<<<grid, block>>>(
			planets->getPositions().map(MEM::MISC::BUF_CU),
			planets->getRadiuses().map(MEM::MISC::BUF_CU),
			clusterer.getCounts()->d_data(),
			clusterer.getShuffle()->d_data(),
			clusterer.getCount() - 1,
			out_merges,
			merge_needed.d_data() );
		CUT_CHECK_ERROR("kernel launch");

		if( merge_needed.retrieve() == 0 )
		{
			if( merge_was_needed )
			{
				TODO("rzadziej filtrować");
				filter.resize(threads);
				create_filter<<<grid, block>>>(
					planets->getMasses().d_data(),
					filter.d_data(),
					planets->getCount().map(MEM::MISC::BUF_CU) );
				planets->filter( &filter );
			}
			return;
		}
		merge_was_needed = true;

	/*	merges2.bind();
		for( unsigned i = 0; i < threads; ++i )
		{
			if( merges2.h_data()[i] != i )
				log_printf( DBG, "merges[%u] = %u\n", i, merges2.h_data()[i] );
		}
		merges2.unbind();
		static unsigned safety_buf = 0;
		if( ++safety_buf > 10 ) abort(); */
		MEM::MISC::BufferCu<unsigned> done(1);
		
		unsigned __debug_counter = 0;
		do
		{
			log_printf(DBG, "mergin' #%u\n", __debug_counter++ );
			if( __debug_counter >= threads )
			{
				PRINT_OUT_BUF( merges1, "%u" );
				PRINT_OUT_BUF( merges2, "%u" );
				abort();
			}
			done.assign(1);
			std::swap( in_merges, out_merges );
			merge_collisions<<<grid, block>>>(
				in_merges,
				out_merges,
				planets->getPositions().map(MEM::MISC::BUF_CU),
				planets->getVelocities().d_data(),
				planets->getMasses().d_data(),
				planets->getRadiuses().map(MEM::MISC::BUF_CU),
				planets->getCount().map(MEM::MISC::BUF_CU),
				done.d_data() );
			CUT_CHECK_ERROR("kernel launch");
		}
		while( !done.retrieve() );
	}
	while(true);
}

void Phx::CImpl::enableClusters(bool orly)
{
	clusters_on = orly;
}

bool Phx::CImpl::clustersEnabled() const
{
	return clusters_on;
}

Phx::Phx(MEM::MISC::PhxPlanetFactory *p)
	: impl( new CImpl(p) )
{
}

Phx::~Phx()
{
	delete impl;
}

void Phx::compute(unsigned n)
{
	impl->compute(n);
}

void Phx::enableClusters(bool orly)
{
	impl->enableClusters(orly);
}

bool Phx::clustersEnabled() const
{
	return impl->clustersEnabled();
}
