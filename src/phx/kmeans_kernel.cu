
#include <hip/hip_runtime.h>

//Random params
const unsigned a = 22695477;
const unsigned c = 1;
const unsigned RANDMAX = (0u - 1u);

__device__ unsigned rand( unsigned & seed )
{
	seed = a * seed + c;
	return seed; 
}

//Float z przedziału [0, max)
__device__ float frand( float max, unsigned & seed)
{
	return (max * rand(seed) ) / RANDMAX;
}

__global__ void kmeans__randomize_kernel(float3* elems, unsigned elems_size, unsigned *seed, float xmax, float ymax, float zmax)
{
	unsigned start = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned my_seed = (666 + start) * (start ^ *seed);
	for(unsigned i = start; i < elems_size ; i += blockDim.x * gridDim.x )
	{
		elems[ i ].x = frand( xmax , my_seed );
		elems[ i ].y = frand( ymax , my_seed );
		elems[ i ].z = frand( zmax , my_seed );
	}
	if(!start)
		*seed = my_seed;
}

__global__ void kmeans__paint_kernel(float3* colors, unsigned color_count, float r, float g, float b)
{
	unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index >= color_count)
		return;
	colors[ index ].x = r;
	colors[ index ].y = g;
	colors[ index ].z = b;
}

// 2 * k < elems_size
__global__ void kmeans__findbest_kernel(unsigned k, float3* means, float3* elems, unsigned elems_size, unsigned* assignments, float* errors)
{
	unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
	extern __shared__ float3 s_means[];
	float x = elems[index].x;
	float y = elems[index].y;
	float z = elems[index].z;

	if( index >= elems_size )
		return;
	if( threadIdx.x < k )//@todo k > 512
		s_means[threadIdx.x] = means[threadIdx.x];

	__syncthreads();
	
	///@todo pozbyć się konfliktów w bankach pamięci?
	unsigned best = 0;
	float dx = s_means[0].x - x;
	float dy = s_means[0].y - y;
	float dz = s_means[0].z - z;
	float dist = dx * dx + dy * dy + dz * dz;
	float best_dist = dist;

	for(unsigned i = 1; i < k; ++i)
	{
		dx = s_means[i].x - x;
		dy = s_means[i].y - y;
		dz = s_means[i].z - z;
		dist = dx * dx + dy * dy + dz * dz;
		if(dist < best_dist)
		{
			best_dist = dist;
			best = i;
		}
	}
	assignments[ index ] = best;
	errors[ index ] = best_dist;
}

__global__ void kmeans__prepare_kernel(unsigned* to_sort, unsigned num)
{
	unsigned index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < num)
		to_sort[index] = index;
}

///Liczy ilości wystąpień wartości nie większych od danej w posortowanej tablicy
__global__ void kmeans__count_kernel(unsigned *assignments, unsigned *counts, unsigned num_minus_1, unsigned k)
{
	unsigned index = threadIdx.x + blockIdx.x * blockDim.x;
	if( index > num_minus_1 )
		return;
	unsigned prev = assignments[ index ];
	unsigned next;
	if( index != num_minus_1 )
		next = assignments[ index + 1 ];
	else
		next = k;
	while( prev != next )
		counts[ prev++ ] = index + 1;
}

///Koloruje klastry
__global__ void kmeans__paint2_kernel(unsigned *assignments, float3* colors, unsigned num, unsigned* shuffle )
{
	unsigned index = threadIdx.x + blockIdx.x * blockDim.x;
	if( index >= num )
		return;
	unsigned index_s = shuffle[ index ];
	colors[ index_s ].x = ((assignments[ index ] + 1) % 3) / 2.0;
	colors[ index_s ].y = ((assignments[ index ]/3 + 1) % 3) / 2.0;
	colors[ index_s ].z = ((assignments[ index ]/9 + 1) % 3) / 2.0;
}

__global__ void kmeans__calcerror_kernel(float* elems, unsigned elems_size, unsigned* assignments, float* means, unsigned k)
{
	
	extern __shared__ float errors[];
}

__device__ float3& operator+=(float3& a, const float3& b)
{
	a.x += b.x;
	a.y += b.y;
	a.z += b.z;
	return a;
}

__device__ float3 operator+(const float3&a, const float3&b)
{
	return make_float3( a.x + b.x, a.y + b.y, a.z + b.z );
}

__device__ float3 operator/(const float3& a, const float& b)
{
	return make_float3( a.x / b, a.y / b, a.z / b );
}

template <class T, unsigned int blockSize>
__device__ void reduce(T *g_idata, unsigned n, unsigned tid, unsigned i, T sdata[])
{
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
	if (tid < 32)
	{
		if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
		if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
		if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
		if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
		if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
		if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
	}
}

template <unsigned int blockSize>
__global__ void reduceSelective(float3 *g_idata, float3 *g_odata, unsigned *counts, unsigned id, unsigned* shuffle)
{
	extern __shared__ float3 sdata[];
	unsigned int n = counts[id];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize*2) + tid + (id ? counts[id-1] : 0);
	unsigned int gridSize = blockSize*gridDim.x;
	sdata[tid] = make_float3(0,0,0);
	while (i < n) { sdata[tid] += g_idata[ shuffle[i] ]; i += gridSize; }
	__syncthreads();
	reduce<float3, blockSize>(g_idata, n, tid, i, sdata);
	if (tid == 0) g_odata[id] = sdata[0] / (n - (id ? counts[id-1] : 0));
}

template <class T, unsigned int blockSize>
__global__ void reduceFull(T *g_idata, T *g_odata, unsigned n)
{
	extern __shared__ T sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize*2) + tid;
	unsigned int gridSize = blockSize*gridDim.x;
	sdata[tid] = (T)0;
	while (i < n) { sdata[tid] += g_idata[ i ]; i += gridSize; }
	__syncthreads();
	reduce<T, blockSize>(g_idata, n, tid, i, sdata);
}
